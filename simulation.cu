
#include <iostream>
#include <cmath>
#include <vector>
#include <fstream>

#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

struct Particle
{
    float3 position;
    float3 velocity;
    float3 force;
    float mass;
    float charge;
};

struct ParticleGroup
{
    std::vector<Particle> particles;
    int numParticles;
};

// FLOAT3 OPERATOR OVERLOADS
__device__ float3 operator-(const float3 &a, const float3 &b)
{
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ float3 operator*(const float3 &a, const float &b)
{
    return make_float3(a.x * b, a.y * b, a.z * b);
}

__device__ float3 operator*(const float &b, const float3 &a)
{
    return make_float3(a.x * b, a.y * b, a.z * b);
}

__device__ float3 operator+(const float3 &a, const float3 &b)
{
    return make_float3((a.x + b.x), (a.y + b.y), (a.z + b.z));
}

//-------------------------------------------------------------------------------
__global__ void calculateForces(
    Particle *particles,
    int numParticles,
    float deltaTime)
{
    // verify that the thread being used does not exceed the number of particles
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles)
    {
        return;
    }

    // obtain reference to current particle
    Particle &p = particles[idx];
    p.force = make_float3(0.0f, 0.0f, 0.0f);

    for (int i = 0; i < numParticles; ++i)
    {
        if (i == idx)
        {
            // skip making the particle interact with itself
            continue;
        }

        // obtain reference to other particles
        Particle &q = particles[i];
        // calculate position difference between the two particles
        float3 diff = q.position - p.position;
        // calculate distance (magnitude) between particles
        float dist = sqrt(diff.x * diff.x + diff.y * diff.y + diff.z * diff.z);
        // obtain the inverse of the distance
        // 1e-5f is added to dist to avoid division by zero
        // in case the particles are extremely close to each other
        float invDist = 1.0f / (dist + 1e-5f);

        // obtain the correct direction and magnitude of the acceleration vector
        // by using the cube of the inverse distance
        float invDist3 = invDist * invDist * invDist;

        // gravitational force calculation
        float force = p.mass * q.mass * invDist3;
        p.force = p.force + (diff * force);

        // electrostatic force (Coulomb's law)
        float k = 8.99e9f; // Coulomb's constant (N⋅m²/C²)
        float forceElectrostatic = k * p.charge * q.charge * invDist3;
        p.force = p.force + (diff * forceElectrostatic);
    }
}

__global__ void integrateParticles(
    Particle *particles,
    int numParticles,
    float deltaTime)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles)
        return;

    Particle &p = particles[idx];

    // Update velocity
    p.velocity = p.velocity + (p.force * (deltaTime / p.mass));

    // Update position
    p.position = p.position + (p.velocity * deltaTime);
}

void saveParticleData(
    const std::vector<ParticleGroup> &particleGroups,
    int step,
    std::ofstream &file)
{
    for (int g = 0; g < particleGroups.size(); ++g)
    {
        const ParticleGroup &group = particleGroups[g];
        for (int i = 0; i < group.numParticles; ++i)
        {
            const Particle &p = group.particles[i];
            float forceMagnitude = sqrt(p.force.x * p.force.x + p.force.y * p.force.y + p.force.z * p.force.z);

            file << step << "," << g << "," << i << ","
                 << p.position.x << "," << p.position.y << "," << p.position.z << ","
                 << p.velocity.x << "," << p.velocity.y << "," << p.velocity.z << ","
                 << p.charge << "," << forceMagnitude << "\n";
        }
    }
}

int main(int argc, char **argv)
{
    const float FEMTOSECOND = 1e-15f; // 1 femtosecond in seconds
    const float ANGSTROM = 1e-10f;    // 1 angstrom in meters

    // SIMULATION CONFIGURATION VALUES
    //-------------------------------------------------------------------------------
    int numGroups = 2;
    int numParticlesPerGroup;
    std::string log_name = "particle_data.csv";

    // ARG PARSING
    //-------------------------------------------------------------------------------
    if (argc != 4)
    {
        std::cerr << "Usage: " << argv[0] << " <num_particles_per_group> <num_steps> <delta_time>" << std::endl;
        return 1;
    }

    // int numGroups = std::stoi(argv[1]);

    if (!std::stoi(argv[1]) >= 100)
    {
        numParticlesPerGroup = 100;
    }
    int numSteps = std::stoi(argv[2]);
    float deltaTime = std::stof(argv[3]) * FEMTOSECOND;

    std::cout << "Configuration received:" << std::endl;
    std::cout << "Number of groups = " << numGroups
              << ", particles per group = " << numParticlesPerGroup << std::endl;
    std::cout << "Number of steps = " << numSteps
              << ", delta time = " << deltaTime << "femtoseconds" << std::endl;

    // PARTICLE CONFIGURATION
    //-------------------------------------------------------------------------------
    std::vector<ParticleGroup> particleGroups(2); // Create two particle groups: electrons and protons

    // electron group
    particleGroups[0].numParticles = numParticlesPerGroup;
    particleGroups[0].particles.resize(numParticlesPerGroup);
    for (int i = 0; i < numParticlesPerGroup; ++i)
    {
        Particle &p = particleGroups[0].particles[i];
        p.position = make_float3(rand() / (float)RAND_MAX * 10.0f * ANGSTROM,
                                 rand() / (float)RAND_MAX * 10.0f * ANGSTROM,
                                 rand() / (float)RAND_MAX * 10.0f * ANGSTROM);
        p.velocity = make_float3(0.0f, 0.0f, 0.0f);
        p.force = make_float3(0.0f, 0.0f, 0.0f);
        p.mass = 9.10938356e-31f; // electron mass (kg)
        p.charge = -1.0f;         // electron charge (atomic units)
    }

    // proton group
    particleGroups[1].numParticles = numParticlesPerGroup;
    particleGroups[1].particles.resize(numParticlesPerGroup);
    for (int i = 0; i < numParticlesPerGroup; ++i)
    {
        Particle &p = particleGroups[1].particles[i];
        p.position = make_float3(rand() / (float)RAND_MAX * 10.0f * ANGSTROM,
                                 rand() / (float)RAND_MAX * 10.0f * ANGSTROM,
                                 rand() / (float)RAND_MAX * 10.0f * ANGSTROM);
        p.velocity = make_float3(0.0f, 0.0f, 0.0f);
        p.force = make_float3(0.0f, 0.0f, 0.0f);
        p.mass = 1.6726219e-27f; // proton mass (kg)
        p.charge = 1.0f;         // proton charge (atomic units)
    }

    // SIMULATION CONFIGURATION
    //-------------------------------------------------------------------------------
    int saveInterval = 100; // save data every 100 steps

    std::cout << "Creating log file: " << log_name << std::endl;
    std::ofstream file(log_name);
    file << "Step,Group,ParticleID,PositionX,PositionY,PositionZ,VelocityX,VelocityY,VelocityZ,Charge,ForceMagnitude\n";

    // SIMULATION LOOP
    //-------------------------------------------------------------------------------
    for (int step = 0; step < numSteps; ++step)
    {
        for (int g = 0; g < numGroups; ++g)
        {
            ParticleGroup &group = particleGroups[g];
            Particle *d_particles;
            hipMalloc(&d_particles, group.numParticles * sizeof(Particle));
            hipMemcpy(d_particles, group.particles.data(), group.numParticles * sizeof(Particle), hipMemcpyHostToDevice);

            calculateForces<<<(group.numParticles + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
                d_particles,
                group.numParticles,
                deltaTime);

            integrateParticles<<<(group.numParticles + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
                d_particles,
                group.numParticles,
                deltaTime);

            hipMemcpy(group.particles.data(), d_particles, group.numParticles * sizeof(Particle), hipMemcpyDeviceToHost);
            hipFree(d_particles);
        }

        if (step % saveInterval == 0)
        {
            saveParticleData(particleGroups, step, file);
        }
    }

    // SIMULATION END
    //-------------------------------------------------------------------------------

    return 0;
}