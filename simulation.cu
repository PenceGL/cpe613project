#include "hip/hip_runtime.h"

#include <iostream>
#include <cmath>
#include <vector>
#include <fstream>
#include <random>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"

#define BLOCK_SIZE 256
#define MAX_FLOAT 3.402823466e+38f
#define FEMTOSECOND 1e-15f // 1 femtosecond in seconds
#define ANGSTROM 1e-10f    // 1 angstrom in meters
#define ANGSTROMSQUARED 1e-20f
#define COLOUMB_CONSTANT 8.987551787e9f // Coulomb's constant (N⋅m^2/C^2)
#define GRAVITY 6.67430e-11

struct Particle
{
    int id;
    float3 position;
    float3 velocity;
    float3 force;
    float mass;
    float charge;
};

// ###############################################################################
__global__ void calculateForces(
    Particle *targetParticles,
    Particle *otherParticles,
    int numParticles,
    float deltaTime)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= numParticles)
    {
        return;
    }

    Particle &target = targetParticles[idx];
    target.force = make_float3(0.0f, 0.0f, 0.0f);

    for (int i{0}; i < numParticles; ++i)
    {
        // obtain reference to each of the other particles
        Particle &other = otherParticles[i];

        // calculate distance between the two particles
        float3 distanceVector = other.position - target.position;
        float distance = length(distanceVector);
        // calculate the unit vector pointing between the objects
        float3 forceDirection = distanceVector / distance;

        // calculate gravitational force
        // F = G * (m1 * m2) / (r^2)
        float gravMagnitude = GRAVITY * (target.mass * other.mass) / (distance * distance);
        // calculate electrostatic force
        float electroMagnitude = COLOUMB_CONSTANT * fabs(target.charge * other.charge) / (distance * distance);
        float3 force = forceDirection * (gravMagnitude + electroMagnitude);

        if (target.charge * other.charge < 0)
        {
            // attractive force
            target.force -= force;
            other.force += force;
        }
        else
        {
            // repulsive force
            target.force += force;
            other.force -= force;
        }
    }
}

__global__ void integrateParticles(
    Particle *particles,
    int numParticles,
    float deltaTime)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= numParticles)
        return;

    Particle &target = particles[idx];

    float3 acceleration = target.force / target.mass;
    target.velocity += (acceleration * deltaTime);
    target.position += (target.velocity * deltaTime);
}

__global__ void saveParticleData(
    const Particle *electrons,
    const Particle *protons,
    int numParticles,
    int step,
    float *distances,
    int *nearestProtonIds)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles)
        return;

    const Particle &electron = electrons[idx];

    float minDistance = MAX_FLOAT;
    int nearestProtonId = -1;

    for (int j = 0; j < numParticles; ++j)
    {
        const Particle &proton = protons[j];

        // calculate distance between the two particles
        float distance = length(proton.position - electron.position);

        if (distance < minDistance)
        {
            minDistance = distance;
            nearestProtonId = proton.id;
        }
    }

    distances[idx] = minDistance;
    nearestProtonIds[idx] = nearestProtonId;
}

int main(int argc, char **argv)
{
    // SIMULATION CONFIGURATION VALUES
    //-------------------------------------------------------------------------------
    int numParticlesPerGroup = 0;
    int numSteps = 0;
    float deltaTime = 0.0;
    std::string log_name = "particle_data.csv";

    // ARGUMENT PARSING
    //-------------------------------------------------------------------------------
    if (argc != 4)
    {
        std::cerr << "Usage: " << argv[0] << " <num_particles_per_group> <num_steps> <delta_time>" << std::endl;
        return 1;
    }

    numParticlesPerGroup = std::stoi(argv[1]);
    // enforce a minimum of at least one particle in each group
    if (numParticlesPerGroup < 1)
    {
        numParticlesPerGroup = 1;
    }

    numSteps = std::stoi(argv[2]);
    // enforce a minimum number of steps
    if (numSteps < 50)
    {
        numSteps = 50;
    }

    deltaTime = std::stof(argv[3]);
    if (deltaTime < 0.001)
    {
        deltaTime = 0.001;
    }
    // convert delta time to femtoseconds
    deltaTime *= FEMTOSECOND;

    std::cout << "Configuration received:" << std::endl;
    std::cout << "-----------------------------------------------------------" << std::endl;
    std::cout << "\tNumber of particles = " << numParticlesPerGroup << std::endl;
    std::cout << "\tNumber of steps = " << numSteps << std::endl;
    std::cout << "\tDelta time per step = " << deltaTime << "femtoseconds" << std::endl;
    std::cout << "-----------------------------------------------------------" << std::endl;

    // PARTICLE CONFIGURATION
    //-------------------------------------------------------------------------------
    // create two particle groups: one for electrons and one for protons
    std::vector<Particle> electrons;
    std::vector<Particle> protons;

    // random number generator
    std::mt19937 rng(std::random_device{}());
    std::uniform_real_distribution<float> posRange(0.0f, 0.1f);
    std::uniform_real_distribution<float> velRange(-0.01f, 0.01f);

    const float BOHR_RADIUS = 0.529177f * ANGSTROM;

    electrons.resize(numParticlesPerGroup);
    for (int i{0}; i < numParticlesPerGroup; ++i)
    {
        Particle &e = electrons[i];
        e.id = i;
        // e.position = make_float3(posRange(rng) * ANGSTROM,
        //                          posRange(rng) * ANGSTROM,
        //                          posRange(rng) * ANGSTROM);
        // e.velocity = make_float3(velRange(rng) * ANGSTROM / FEMTOSECOND,
        //                          velRange(rng) * ANGSTROM / FEMTOSECOND,
        //                          velRange(rng) * ANGSTROM / FEMTOSECOND);
        // e.force = make_float3(0.0f, 0.0f, 0.0f);

        e.position = make_float3(BOHR_RADIUS, 0.0f, 0.0f);
        e.velocity = make_float3(0.0f, 0.0f, 0.0f);
        e.force = make_float3(0.0f, 0.0f, 0.0f);

        e.mass = 9.10938356e-31f;    // electron mass (kg)
        e.charge = -1.602176634e-19; // Charge of electron (Coulombs)

        std::cout << "Electron " << i << " initial position = "
                  << e.position.x << ", "
                  << e.position.y << ", "
                  << e.position.z << std::endl;
    }

    protons.resize(numParticlesPerGroup);
    for (int i{0}; i < numParticlesPerGroup; ++i)
    {
        Particle &p = protons[i];
        p.id = i;
        // p.position = make_float3(posRange(rng) * ANGSTROM,
        //                          posRange(rng) * ANGSTROM,
        //                          posRange(rng) * ANGSTROM);
        // p.velocity = make_float3(velRange(rng) * ANGSTROM / FEMTOSECOND,
        //                          velRange(rng) * ANGSTROM / FEMTOSECOND,
        //                          velRange(rng) * ANGSTROM / FEMTOSECOND);
        // p.force = make_float3(0.0f, 0.0f, 0.0f);

        p.position = make_float3(0.0f, 0.0f, 0.0f);
        p.velocity = make_float3(0.0f, 0.0f, 0.0f);
        p.force = make_float3(0.0f, 0.0f, 0.0f);

        p.mass = 1.6726219e-27f;    // proton mass (kg)
        p.charge = 1.602176634e-19; // Charge of proton (Coulombs)

        std::cout << "Proton " << i << " initial position = "
                  << p.position.x << ", "
                  << p.position.y << ", "
                  << p.position.z << std::endl;
    }

    // float3 positionVector = electrons[0].position - protons[0].position;
    // float3 velocityDirection = make_float3(-positionVector.y, positionVector.x, 0.0f);
    // velocityDirection = normalize(velocityDirection);
    // electrons[0].velocity = velocityDirection * (1.0f * ANGSTROM / FEMTOSECOND);

    float distanceX = protons[0].position.x - electrons[0].position.x;
    float distanceY = protons[0].position.y - electrons[0].position.y;
    float distanceZ = protons[0].position.z - electrons[0].position.z;

    float distanceMagnitude = sqrt(pow(distanceX, 2) + pow(distanceY, 2) + pow(distanceZ, 2));
    std::cout << "Initial distance between particles = " << distanceMagnitude << std::endl;
    std::cout << "Initial position difference between particles (X, Y, Z) = "
              << distanceX << ", "
              << distanceY << ", "
              << distanceZ << std::endl;

    // LOG FILE SETUP
    //-------------------------------------------------------------------------------
    // save interval in number of time steps
    // data will be logged to the output file at increments of this value
    int logInterval = 100;

    std::cout << "Creating log file: " << log_name << std::endl;
    std::ofstream file(log_name);
    file << "Step,ElectronID,NearestProtonID,Distance,ElectronPosX,ElectronPosY,ElectronPosZ,NearestProtonPosX,NearestProtonPosY,NearestProtonPosZ\n";

    // vectors that will be used to store distances between particles and the nearest proton for a given electron
    std::vector<float> distances(numParticlesPerGroup);
    std::vector<int> nearestProtonIds(numParticlesPerGroup);

    // DEVICE MEMORY SETUP
    //-------------------------------------------------------------------------------
    // allocate device memory for particle groups
    Particle *d_electrons;
    Particle *d_protons;
    checkCudaErrors(hipMalloc(&d_electrons, numParticlesPerGroup * sizeof(Particle)));
    checkCudaErrors(hipMalloc(&d_protons, numParticlesPerGroup * sizeof(Particle)));

    // copy particle data from host to device
    checkCudaErrors(hipMemcpy(d_electrons, electrons.data(), numParticlesPerGroup * sizeof(Particle), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_protons, protons.data(), numParticlesPerGroup * sizeof(Particle), hipMemcpyHostToDevice));

    // allocate device memory for output arrays
    float *d_distances;
    int *d_nearestProtonIds;
    checkCudaErrors(hipMalloc(&d_distances, numParticlesPerGroup * sizeof(float)));
    checkCudaErrors(hipMalloc(&d_nearestProtonIds, numParticlesPerGroup * sizeof(int)));

    // SIMULATION LOOP
    //-------------------------------------------------------------------------------
    std::cout << "Launching simulation..." << std::endl;

    int blockDim = BLOCK_SIZE;
    int gridDim = (numParticlesPerGroup + BLOCK_SIZE - 1) / BLOCK_SIZE;

    for (int step = 0; step < numSteps; ++step)
    {
        // launch the saveParticleData kernel at log intervals
        if (step % logInterval == 0)
        {
            saveParticleData<<<gridDim, blockDim>>>(
                d_electrons,
                d_protons,
                numParticlesPerGroup,
                step,
                d_distances,
                d_nearestProtonIds);

            checkCudaErrors(hipMemcpy(distances.data(), d_distances, numParticlesPerGroup * sizeof(float), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(nearestProtonIds.data(), d_nearestProtonIds, numParticlesPerGroup * sizeof(int), hipMemcpyDeviceToHost));

            // write the data to the log file
            for (int i{0}; i < numParticlesPerGroup; ++i)
            {
                const Particle &electron = electrons[i];
                const Particle &proton = protons[nearestProtonIds[i]];

                file << step << "," << electron.id << "," << proton.id << ","
                     << distances[i] << ","
                     << electron.position.x << "," << electron.position.y << "," << electron.position.z << ","
                     << proton.position.x << "," << proton.position.y << "," << proton.position.z << "\n";
            }
        }

        calculateForces<<<gridDim, blockDim>>>(
            d_electrons,
            d_protons,
            numParticlesPerGroup,
            deltaTime);

        calculateForces<<<gridDim, blockDim>>>(
            d_protons,
            d_electrons,
            numParticlesPerGroup,
            deltaTime);

        // calculate all forces prior to integrating
        checkCudaErrors(hipDeviceSynchronize());

        integrateParticles<<<gridDim, blockDim>>>(
            d_electrons,
            numParticlesPerGroup,
            deltaTime);

        integrateParticles<<<gridDim, blockDim>>>(
            d_protons,
            numParticlesPerGroup,
            deltaTime);

        // integrate all particles prior to logging
        checkCudaErrors(hipDeviceSynchronize());

        // copy the updated particle data back to the host after integrating
        checkCudaErrors(hipMemcpy(electrons.data(), d_electrons, numParticlesPerGroup * sizeof(Particle), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(protons.data(), d_protons, numParticlesPerGroup * sizeof(Particle), hipMemcpyDeviceToHost));
    }

    // SIMULATION TEARDOWN
    //-------------------------------------------------------------------------------
    file.close();
    // free device memory
    checkCudaErrors(hipFree(d_electrons));
    checkCudaErrors(hipFree(d_protons));
    checkCudaErrors(hipFree(d_distances));
    checkCudaErrors(hipFree(d_nearestProtonIds));

    std::cout << "Simulation completed successfully." << std::endl;
    return 0;
}