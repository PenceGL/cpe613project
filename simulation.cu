#include "hip/hip_runtime.h"

#include <iostream>
#include <cmath>
#include <vector>
#include <fstream>
#include <random>
#include <iomanip> // for std::setprecision

#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"

#define BLOCK_SIZE 256
#define MAX_FLOAT 3.402823466e+38f      // used for distance comparisons
#define FEMTOSECOND 1e-15f              // 1 femtosecond in seconds
#define ANGSTROM 1e-10f                 // 1 angstrom in meters
#define COULOMB_CONSTANT 8.987551787e9f // Coulomb's constant (N⋅m^2/C^2)
#define GRAVITY 6.67430e-11             // gravitational constant (N⋅m^2⋅kg^−2)
#define BOHR_RADIUS 0.529177f * ANGSTROM

struct Particle
{
    int id;
    float3 position;
    float3 velocity;
    float3 force;
    float mass;
    float charge;
};

__device__ void calculateForces(
    int32_t idx,
    Particle *targets,
    Particle *others,
    int numParticles,
    Particle *sharedParticles)
{
    for (int i{0}; i < numParticles; i += blockDim.x)
    {
        int loadIdx = (i + threadIdx.x);
        if (loadIdx < numParticles)
        {
            sharedParticles[threadIdx.x] = others[loadIdx];
        }
        // sync to ensure all particles have been loaded by all threads
        __syncthreads();

        for (int j{0}; j < blockDim.x && (i + j) < numParticles; ++j)
        {
            Particle &target = targets[idx];
            target.force = make_float3(0.0f, 0.0f, 0.0f);

            // obtain reference to each of the other particles
            Particle &other = sharedParticles[j];

            // calculate vector pointing from the target to the other particle
            float3 distanceVector = other.position - target.position;
            // obtain the magnitude of the distance vector
            float distance = length(distanceVector);
            // divide to obtain the unit vector pointing from the target to the other
            float3 forceDirection = distanceVector / distance;

            // calculate gravitational force
            // F = G * (m1 * m2) / (r^2)
            // float gravMagnitude = GRAVITY * (target.mass * other.mass) / (distance * distance);

            // calculate electrostatic force
            float electroMagnitude = COULOMB_CONSTANT * (fabs(target.charge * other.charge) / (distance * distance));

            // apply the effects of both gravitational and electrostatic forces
            // target.force = forceDirection * (gravMagnitude + electroMagnitude);
            target.force += forceDirection * electroMagnitude;
        }
        // ensure all threads have finished using shared memory before next load
        __syncthreads();
    }
}

__device__ void integrateMotion(
    int idx,
    Particle *particles,
    float deltaTime)
{
    Particle &target = particles[idx];

    float3 acceleration = target.force / target.mass;
    target.velocity += (acceleration * deltaTime);
    target.position += (target.velocity * deltaTime);
}

// ###############################################################################
__global__ void simulationStep(
    Particle *electrons,
    Particle *protons,
    int numParticles,
    float deltaTime)
{
    extern __shared__ Particle sharedParticles[];
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numParticles)
    {
        calculateForces(idx, electrons, protons, numParticles, sharedParticles);
        calculateForces(idx, protons, electrons, numParticles, sharedParticles);
        // synchronize to ensure all forces are updated before modifying motion of particles
        __syncthreads();
        integrateMotion(idx, electrons, deltaTime);
        integrateMotion(idx, protons, deltaTime);
    }
}

__global__ void findNearestProton(
    const Particle *electrons,
    const Particle *protons,
    int numParticles,
    int step,
    float *distances,
    int *nearestProtonIds)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numParticles)
        return;

    const Particle &electron = electrons[idx];

    float nearestProtonDistance = MAX_FLOAT;
    int32_t nearestProtonId = -1;

    for (int j{0}; j < numParticles; ++j)
    {
        const Particle &proton = protons[j];

        float3 posDiff = proton.position - electron.position;
        float distance = length(posDiff);

        if (distance < nearestProtonDistance)
        {
            nearestProtonDistance = distance;
            nearestProtonId = proton.id;
        }
    }

    distances[idx] = nearestProtonDistance;
    nearestProtonIds[idx] = nearestProtonId;
}

void hostPrintParticleData(
    const std::vector<Particle> &electrons,
    const std::vector<Particle> &protons,
    int numParticles,
    int step,
    const std::vector<float> &distances,
    const std::vector<int> &nearestProtonIds)
{
    std::cout << "=================== Step " << step << " ===================" << std::endl;
    for (int i = 0; i < numParticles; ++i)
    {
        const Particle &electron = electrons[i];
        const Particle &proton = protons[i];

        float distanceX = proton.position.x - electron.position.x;
        float distanceY = proton.position.y - electron.position.y;
        float distanceZ = proton.position.z - electron.position.z;

        float distance = sqrt(pow(distanceX, 2) + pow(distanceY, 2) + pow(distanceZ, 2));

        std::cout << std::scientific << std::setprecision(3);

        std::cout << "Electron " << electron.id << ":" << std::endl
                  << "\tpos[" << electron.position.x << ", " << electron.position.y << ", " << electron.position.z << "]" << std::endl
                  << "\tvel[" << electron.velocity.x << ", " << electron.velocity.y << ", " << electron.velocity.z << "]" << std::endl
                  << "\tfrc[" << electron.force.x << ", " << electron.force.y << ", " << electron.force.z << "]" << std::endl;

        std::cout
            << "Proton " << proton.id << ":" << std::endl
            << "\tpos[" << proton.position.x << ", " << proton.position.y << ", " << proton.position.z << "]" << std::endl
            << "\tvel[" << proton.velocity.x << ", " << proton.velocity.y << ", " << proton.velocity.z << "]" << std::endl
            << "\tfrc[" << proton.force.x << ", " << proton.force.y << ", " << proton.force.z << "]" << std::endl;

        std::cout << "proton frcDir[" << distanceX / distance << ", " << distanceY / distance << ", " << distanceZ / distance << "]" << std::endl;
        std::cout << "dist = " << distance << std::endl;
    }
    std::cout << "===============================================" << std::endl;
}

int main(int argc, char **argv)
{
    // SIMULATION CONFIGURATION VALUES
    //-------------------------------------------------------------------------------
    int numParticlesPerGroup = 0;
    int numSteps = 0;
    float deltaTime = 0.0;
    std::string log_name = "particle_data.csv";

    // ARGUMENT PARSING
    //-------------------------------------------------------------------------------
    if (argc != 4)
    {
        std::cerr << "Usage: " << argv[0] << " <num_particles_per_group> <num_steps> <delta_time>" << std::endl;
        return 1;
    }

    numParticlesPerGroup = std::stoi(argv[1]);
    // enforce a minimum of at least one particle in each group
    if (numParticlesPerGroup < 1)
    {
        numParticlesPerGroup = 1;
    }

    numSteps = std::stoi(argv[2]);
    // enforce a minimum number of steps
    if (numSteps < 20)
    {
        numSteps = 20;
    }

    deltaTime = std::stof(argv[3]);
    if (deltaTime > 0.001)
    {
        std::cout << "Provided time step is too large and will cause loss of simulation "
                  << "fidelity. Reverting to default of 0.001 femtoseconds." << std::endl;
        deltaTime = 0.001;
    }
    // convert delta time to femtoseconds
    deltaTime *= FEMTOSECOND;

    std::cout << "Configuration received:" << std::endl;
    std::cout << "-----------------------------------------------------------" << std::endl;
    std::cout << "\tNumber of particles in each group = " << numParticlesPerGroup << std::endl;
    std::cout << "\tNumber of steps = " << numSteps << std::endl;
    std::cout << "\tDelta time per step = " << deltaTime << " seconds" << std::endl;
    std::cout << "-----------------------------------------------------------" << std::endl;

    // PARTICLE INSTANTIATION/CONFIGURATION
    //-------------------------------------------------------------------------------
    // create two particle groups: one for electrons and one for protons
    std::vector<Particle> electrons;
    std::vector<Particle> protons;

    // random number generator
    std::mt19937 rng(std::random_device{}());
    std::uniform_real_distribution<float> posRange(0.0f, 0.1f);
    std::uniform_real_distribution<float> velRange(-0.01f, 0.01f);

    electrons.resize(numParticlesPerGroup);
    for (int i{0}; i < numParticlesPerGroup; ++i)
    {
        Particle &e = electrons[i];
        e.id = i;
        e.position = make_float3(posRange(rng) * ANGSTROM,
                                 posRange(rng) * ANGSTROM,
                                 posRange(rng) * ANGSTROM);
        e.velocity = make_float3(velRange(rng) * ANGSTROM / FEMTOSECOND,
                                 velRange(rng) * ANGSTROM / FEMTOSECOND,
                                 velRange(rng) * ANGSTROM / FEMTOSECOND);
        e.force = make_float3(0.0f, 0.0f, 0.0f);

        // e.position = make_float3(BOHR_RADIUS, 0.0f, 0.0f);
        // e.velocity = make_float3(0.0f, 0.0f, 0.0f);
        // e.force = make_float3(0.0f, 0.0f, 0.0f);

        e.mass = 9.10938356e-31f;    // electron mass (kg)
        e.charge = -1.602176634e-19; // Charge of electron (Coulombs)
    }

    protons.resize(numParticlesPerGroup);
    for (int i{0}; i < numParticlesPerGroup; ++i)
    {
        Particle &p = protons[i];
        p.id = i;
        p.position = make_float3(posRange(rng) * ANGSTROM,
                                 posRange(rng) * ANGSTROM,
                                 posRange(rng) * ANGSTROM);
        p.velocity = make_float3(velRange(rng) * ANGSTROM / FEMTOSECOND,
                                 velRange(rng) * ANGSTROM / FEMTOSECOND,
                                 velRange(rng) * ANGSTROM / FEMTOSECOND);
        p.force = make_float3(0.0f, 0.0f, 0.0f);

        // p.position = make_float3(0.0f, 0.0f, 0.0f);
        // p.velocity = make_float3(0.0f, 0.0f, 0.0f);
        // p.force = make_float3(0.0f, 0.0f, 0.0f);

        p.mass = 1.6726219e-27f;    // proton mass (kg)
        p.charge = 1.602176634e-19; // Charge of proton (Coulombs)
    }

    // TEMP: apply an initial velocity to the electron that causes it to orbit the proton
    // float electron_charge = 1.602176634e-19; // electron charge in Coulombs
    // float electron_mass = 9.10938356e-31;    // electron mass in kilograms
    // float r = BOHR_RADIUS;
    // // Velocity for circular orbit at the Bohr radius
    // float v = sqrt((COULOMB_CONSTANT * electron_charge * electron_charge) / (electron_mass * r));
    // // Set the electron's initial velocity to be perpendicular to the radius vector
    // // Assuming the proton is at the origin and the electron is at position (BOHR_RADIUS, 0, 0)
    // electrons[0].velocity = make_float3(0.0f, v, 0.0f);

    // LOG FILE SETUP
    //-------------------------------------------------------------------------------
    // save interval in number of time steps
    // data will be logged to the output file at increments of this value
    int logInterval = 25;

    std::cout << "Creating log file: " << log_name << std::endl;
    std::ofstream file(log_name);
    file << "Step,ElectronID,NearestProtonID,Distance,ElectronPosX,ElectronPosY,ElectronPosZ,NearestProtonPosX,NearestProtonPosY,NearestProtonPosZ\n";

    // vectors that will be used to store distances between particles and the nearest proton for a given electron
    std::vector<float> distances(numParticlesPerGroup);
    std::vector<int> nearestProtonIds(numParticlesPerGroup);

    // DEVICE MEMORY SETUP
    //-------------------------------------------------------------------------------
    size_t particleMem = numParticlesPerGroup * sizeof(Particle);
    size_t floatMem = numParticlesPerGroup * sizeof(float);
    size_t intMem = numParticlesPerGroup * sizeof(int32_t);

    // allocate device memory for particle groups
    Particle *d_electrons;
    Particle *d_protons;
    float *d_distances;
    int32_t *d_nearestProtonIds;

    // malloc for particle array input
    checkCudaErrors(hipMalloc(&d_electrons, particleMem));
    checkCudaErrors(hipMalloc(&d_protons, particleMem));

    // copy particle data to device
    checkCudaErrors(hipMemcpy(d_electrons, electrons.data(), particleMem, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_protons, protons.data(), particleMem, hipMemcpyHostToDevice));

    // malloc/memset for distances output
    checkCudaErrors(hipMalloc(&d_distances, floatMem));
    checkCudaErrors(hipMemset(d_distances, 0, floatMem));
    // malloc/memset for proton IDs output
    checkCudaErrors(hipMalloc(&d_nearestProtonIds, intMem));
    checkCudaErrors(hipMemset(d_nearestProtonIds, 0, intMem));

    int blockDim = BLOCK_SIZE;
    int gridDim = (numParticlesPerGroup + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int sharedMemSize = sizeof(Particle) * BLOCK_SIZE;

    // SIMULATION LOOP
    //-------------------------------------------------------------------------------
    std::cout << "Launching simulation..." << std::endl;

    hipEvent_t cudaStartEvent, cudaStopEvent;
    checkCudaErrors(hipEventCreate(&cudaStartEvent));
    checkCudaErrors(hipEventCreate(&cudaStopEvent));

    checkCudaErrors(hipEventRecord(cudaStartEvent));

    for (uint32_t step{0}; step < numSteps; ++step)
    {
        // launch the findNearestProton kernel at log intervals
        if (step % logInterval == 0)
        {
            findNearestProton<<<gridDim, blockDim>>>(
                d_electrons,
                d_protons,
                numParticlesPerGroup,
                step,
                d_distances,
                d_nearestProtonIds);

            // copy all particle data back to host for logging
            checkCudaErrors(hipMemcpy(electrons.data(), d_electrons, particleMem, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(protons.data(), d_protons, particleMem, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(distances.data(), d_distances, floatMem, hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy(nearestProtonIds.data(), d_nearestProtonIds, intMem, hipMemcpyDeviceToHost));

            // write the resulting data to the log file at each log interval
            for (int i{0}; i < numParticlesPerGroup; ++i)
            {
                const Particle &electron = electrons[i];
                const Particle &proton = protons[nearestProtonIds[i]];
                file << std::scientific << std::setprecision(3)
                     << step << "," << electron.id << "," << proton.id << ","
                     << distances[i] << ","
                     << electron.position.x << "," << electron.position.y << "," << electron.position.z << ","
                     << proton.position.x << "," << proton.position.y << "," << proton.position.z << "\n";
            }
        }

        simulationStep<<<gridDim, blockDim, sharedMemSize>>>(
            d_electrons,
            d_protons,
            numParticlesPerGroup,
            deltaTime);
    }

    checkCudaErrors(hipEventRecord(cudaStopEvent));
    checkCudaErrors(hipEventSynchronize(cudaStopEvent));

    float milliseconds = 0;
    checkCudaErrors(hipEventElapsedTime(&milliseconds, cudaStartEvent, cudaStopEvent));
    std::cout << std::defaultfloat << "Simulation duration = " << milliseconds << "ms" << std::endl;

    // SIMULATION TEARDOWN
    //-------------------------------------------------------------------------------
    file.close();
    // free device memory
    checkCudaErrors(hipFree(d_electrons));
    checkCudaErrors(hipFree(d_protons));
    checkCudaErrors(hipFree(d_distances));
    checkCudaErrors(hipFree(d_nearestProtonIds));

    std::cout << "Simulation teardown complete." << std::endl;
    return 0;
}